
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>



//Kernel definition
template<typename T>
__global__
void oftKernel (T* out,
		T* in,
		const unsigned int sd_size,
		const unsigned int block_size,
		const unsigned int I,
		const unsigned int L)
{
	const unsigned int sd_id = static_cast<int> (threadIdx.x / L); //automatically rounded down in int arithmetics
	const unsigned int id = threadIdx.x - sd_id * L;
	const unsigned int sd_start = blockIdx.x * blockDim.x * I + sd_id * L * I;
	
	for (unsigned int i = 0; i < I; i++)
	{
		const unsigned el_id = sd_start + i * L + id;
		((T*) out)[el_id] = ((T*) in)[el_id];
	//	out[el_id] = in[el_id]; 
//		((T*) out)[0] = ((T*) in)[0];
	}
}

int main () {
	using namespace std::chrono;

	std::cout << "np.array("; //output the results so that they can be read easily by python
	
	std::cout << "(";
	for (int n = 0; n <= 5; n++)
	{			
		std::cout << "(";
		for(int j = 0; j <= 10; j++)
		{
			unsigned int I = 1 << j;
			unsigned int size = 1 << 30;
			unsigned int L = 4;
			unsigned int N = 8 * (1<<n);
			unsigned int sd_size;
			int Tindx = 5;
			switch(Tindx)
			{
				case 1 :
					sd_size = I * L * sizeof(char);
					break;
				case 2 :
					sd_size = I * L * sizeof(short);
					break;
				case 3 :
					sd_size = I * L * sizeof(int);
					break;
				case 4 :
					sd_size = I * L * sizeof(int2);
					break;
				case 5 :
					sd_size = I * L * sizeof(int4);
					break;
			}
			
			unsigned int block_size = sd_size * N;
			unsigned int block_amount = size / block_size; 
		
			void* out;
			void* in;
			
			auto err1 = hipMalloc(&out, block_size * block_amount);
			auto err2 = hipMalloc(&in, block_size * block_amount);	
  
			//initArrays
			hipMemset(in, 111, size);
			hipMemset(out, 4, size);			
	
	//		size_t free;
	//		size_t total;
	//		auto err3 = cudaMemGetInfo(&free, &total);

                  	if (err2 != hipSuccess)
			{	
                        	std::cout << "ERROR: " << hipGetErrorString(err2) << std::endl;
  			}
	//		for (int x = 1; x <= 10; x++) {
	//			oftKernel<<<block_amount, L * N >>> (out, in, sd_size, block_size, I, L);	
	//			cudaDeviceSynchronize();
	//		}
	//		std::cout<<"free:" <<free << " total:" << total << " savedArrays: " << (total - free)/ (block_size * block_amount) << " j:" << j << " Tindx:" << Tindx << std::endl;		
		
	//		cudaFree(out);
	//		cudaFree(in);
			
			//make a warmup 
			switch(Tindx)
			{
				case 1 :
					oftKernel<<<block_amount, L * N >>> (static_cast<char*> (out), static_cast<char*> (in), sd_size, block_size, I, L);
					break;
				case 2 :
					oftKernel<<<block_amount, L * N >>> (static_cast<short*> (out), static_cast<short*> (in), sd_size, block_size, I, L);
					break;
				case 3 :
					oftKernel<<<block_amount, L * N >>> (static_cast<int*> (out), static_cast<int*> (in), sd_size, block_size, I, L);
					break;
				case 4 :
					oftKernel<<<block_amount, L * N >>> (static_cast<int2*> (out), static_cast<int2*> (in), sd_size, block_size, I, L);
					break;
				case 5 :
					oftKernel<<<block_amount, L * N >>> (static_cast<int4*> (out), static_cast<int4*> (in), sd_size, block_size, I, L);
					break;
			}
	
			hipDeviceSynchronize();

			//Time Measururement Point 1
			high_resolution_clock::time_point timeBefore = high_resolution_clock::now();

			for(int x = 1; x <= 100; x++)//run 100 times for better measurement accuracy
			{
				switch(Tindx)
				{
					case 1 :
						oftKernel<<<block_amount, L * N >>> (static_cast<char*> (out), static_cast<char*> (in), sd_size, block_size, I, L);
						break;
					case 2 :
						oftKernel<<<block_amount, L * N >>> (static_cast<short*> (out), static_cast<short*> (in), sd_size, block_size, I, L);
						break;
					case 3 :
						oftKernel<<<block_amount, L * N >>> (static_cast<int*> (out), static_cast<int*> (in), sd_size, block_size, I, L);
						break;
					case 4 :
						oftKernel<<<block_amount, L * N >>> (static_cast<int2*> (out), static_cast<int2*> (in), sd_size, block_size, I, L);
						break;
					case 5 :
						oftKernel<<<block_amount, L * N >>> (static_cast<int4*> (out), static_cast<int4*> (in), sd_size, block_size, I, L);
						break;
				
				}
				
				hipDeviceSynchronize();
			
				auto lstErr = hipGetLastError();
				if ( hipSuccess != lstErr )
				{
					std::cout <<"runningError:"<< lstErr  << ": " << hipGetErrorString(lstErr)  << std::endl;
				}
			}
	//			oftKernel<<<block_amount, L * N >>> (out, in, sd_size, block_size, I, L);
				
		//		std::cout<< "size of out:" << sizeof(out)  <<  "tindx:" << Tindx << " block_amount:" << block_amount << " L:" << L << " N:" << N << " block_size: " << block_size  <<  std::endl;	

			//	cudaDeviceSynchronize();	
			//	oftKernel<<<block_amount, L * N >>> (static_cast<int4*> (out), static_cast<int4*> (in), sd_size, block_size, I, L);
		//		cudaDeviceSynchronize();
						
			
			//Time Measurement Point 2
			high_resolution_clock::time_point timeAfter = high_resolution_clock::now();			
		
			//Output Time Measurement Result
			duration<double> time_span = duration_cast<duration<double>>(timeAfter - timeBefore);
			
			std::cout << time_span.count();
	
			//Check for copy errors
			void* checkAry = malloc(size);
			hipMemcpy(checkAry, out, size , hipMemcpyDeviceToHost);

			for (int pos = 0; pos < size; pos++)
			{
                        	if (static_cast<char*> (checkAry)[pos] != 111)
			{
                                	std::cout << "Copy Misstake at:" << pos <<"with:"<< static_cast<int>(static_cast<char*> (checkAry)[pos])
                                	 << "instead of:" <<static_cast<int>( 111) << std::endl;
                                }
 			 }

	
			hipFree(out);
			hipFree(in);
			free(checkAry);
		
			if( j != 10) {std::cout << ",";} //output a , if we aren't the last element of the for loop	
		}
		
		std::cout << ")";

				
		if( n != 5) {std::cout << ",";} //output a , if we aren't the last element of the for loop	
	}
	
	std::cout << ")";
	
	std::cout << ")" << std::endl;


			
	return 0;
}


